
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 64
#define TPB 32 // thread per block

__device__ float scale(int i, int n){
  return static_cast<float>(i)/(n-1);
}

// Compute the distance between 2 points in a line
__device__ float distance(float x1, float x2){
  return sqrt((x2-x1)*(x2-x1));
}

__global__ void distanceKernel(float *d_out, float ref, int len){
  const int i= blockIdx.x*blockDim.x + threadIdx.x;
  const float x = scale(i, len);
  d_out[i] = distance(x, ref);
  // printf("i=%2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

int main(){
  float *d_out = 0;

  const float ref = 0.5f;

  hipMalloc(&d_out, N*sizeof(float));

  // block num: N/TPB
  distanceKernel<<<N/TPB, TPB>>>(d_out, ref,  N);

  hipFree(d_out);
  return 0;
}