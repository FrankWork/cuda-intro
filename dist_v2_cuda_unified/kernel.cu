
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 64
#define TPB 32 // thread per block

float scale(int i, int n){
  return static_cast<float>(i)/(n-1);
}

// Compute the distance between 2 points in a line
__device__ 
float distance(float x1, float x2){
  return sqrt((x2-x1)*(x2-x1));
}

__global__ 
void distanceKernel(float *d_out, float *d_in, float ref){
  const int i= blockIdx.x*blockDim.x + threadIdx.x;
  const float x = d_in[i];
  d_out[i] = distance(x, ref);
  // printf("i=%2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

int main(){
  float *in = 0;
  float *out = 0;

  const float ref = 0.5f;

  hipMallocManaged(&in, N*sizeof(float));
  hipMallocManaged(&out, N*sizeof(float));

  for(int i=0;i<N;++i){
    in[i] = scale(i,N);
  }

  // block num: N/TPB
  distanceKernel<<<N/TPB, TPB>>>(out, in, ref);
  hipDeviceSynchronize();

  hipFree(in);
  hipFree(out);
  return 0;
}